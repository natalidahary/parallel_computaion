#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "myProto.h"



/*performing a smooth oscillatory motion between two points (x1 and x2) over time (t). 
It calculates the position x at a given time t using the sine function, which results in a periodic 
back-and-forth movement between the two points. */
__device__ double calculateX(double x1, double x2, double t)
{
/*
*param x1 (double): The initial position of the first point.
*param x2 (double): The initial position of the second point.
*param t (double): The time value for which the x value is to be calculated.
**/
    return ((x2 - x1) / 2) * sin(t * M_PI / 2) + ((x2 + x1) / 2);
}


//performing a linear transformation of the 'x' value to calculate the corresponding 'y' value based on the equation y = ax + b. 
__device__ double calculateY(double a, double b, double x)
{
/*
*param a (double): The coefficient 'a' in the linear equation y = ax + b.
*param b (double): The constant term 'b' in the linear equation y = ax + b.
*param x (double): The 'x' value for which the 'y' value is to be calculated.
**/
    return a * x + b;
}


/*
computing the Euclidean distance between two points (p1 and p2) in a 2D space at a given time t. 
It combines calculations from the calculateX and calculateY functions to determine the positions of the points 
and then calculates the distance between them using the Euclidean distance formula
*/
__device__ double calculateDistance(const Point *p1, const Point *p2, double t)
{
/*
*param p1 (const Point*): A pointer to the first Point structure representing the initial state of the first point.
*param p2 (const Point*): A pointer to the second Point structure representing the initial state of the second point.
*param t (double): The time value for which the distance between the points is to be calculated.
**/
    double x1 = calculateX(p1->x1, p1->x2, t);
    double y1 = calculateY(p1->a, p1->b, x1);

    double x2 = calculateX(p2->x1, p2->x2, t);
    double y2 = calculateY(p2->a, p2->b, x2);

    //Calculate the Euclidean distance between the two points (p1 and p2) at time t.
    return sqrt(pow(x2 - x1, 2) + pow(y2 - y1, 2));
}



//This device function checks whether the proximity criteria between two points p1 and p2 are met at a given time t and distance threshold D.
//Return: A boolean value indicating whether the proximity criteria are met.
__device__ bool isProximityCriteriaMet(const Point *p1, const Point *p2, double t, double D)
{
    /**
    *param p1: A pointer to the first Point structure representing one of the points.
    *param p2: A pointer to the second Point structure representing the other point.
    *param t: A double representing the time value.
    *param D: A double representing the distance threshold.
    **/

    //Get the id of the first point (p1) to avoid comparing the point with itself
    int currentPointIndex = p1->id;
  
    //Check if the points are the same (id of p1 and p2 are equal). 
    //If so, return false since a point is not considered in proximity to itself
    if (currentPointIndex == p2->id)
        return false;
    //Calculate the distance between the two points using the calculateDistance function
    double distance = calculateDistance(p1, p2, t);
    //Check if the calculated distance is less than or equal to the threshold D. If it is, return true, 
    //indicating that the proximity criteria are met. Otherwise, return false
    return distance <= D;
}



/*This device function updates the results array with the proximityPointId value for a specific time point index idx and constraint index
To avoid the race condition, you can use atomic operations to perform a compare-and-swap (CAS) operation. 
The CAS operation will only update the element if its value is still equal to -1, 
ensuring that only one thread successfully updates the result for each time point and constraint index.*/
__device__ void updateResults(int idx, int *results, int proximityPointId)
{
    /**
    *param idx: An integer representing the time point index.
    *param results: A pointer to the results array.
    *param proximityPointId: An integer representing the ID of the point that satisfies the proximity 
    **/

    //Loop through the constraints (indexed by j)
    for (int j = 0; j < CONSTRAINTS; j++)
    {
        // Calculate the index of the element to update in the results array based on the current idx and the constraint index j.
        int targetIndex = idx * CONSTRAINTS + j;
        // Use an atomic compare-and-swap (CAS) operation to update the value at targetIndex in the results array.
        // The CAS operation ensures that only one thread successfully updates the result for each time point and constraint index.
        int oldValue = -1;
        int newValue = proximityPointId;
        if (atomicCAS(&results[targetIndex], oldValue, newValue) == oldValue)
        {
            // The CAS operation succeeded, meaning this thread has updated the result successfully.
            // We can break out of the loop to avoid unnecessary checks.
            break;
        }
    }
}


/*This device function calculates the number of points in proximity to the point at currentPointIndex at a given time t and distance threshold D.
Return: An integer representing the number of points in proximity*/
__device__ int countProximityPoints(const Point *points, int N, int currentPointIndex, double t, double D)
{
/*
*param points (const Point*): A pointer to the array of Point structures representing the initial state of the points.
*param N (int): The total number of points in the array.
*param currentPointIndex (int): The index of the point for which proximity points are to be counted.
*param t (double): The time value for which the proximity criteria are to be checked.
*param D (double): The threshold distance used as the proximity criteria.
**/
    //Initialize a counter variable count to 0
    int count = 0;
    //Loop through all the points (j) to check their proximity to the point at currentPointIndex
    for (int j = 0; j < N; j++)
    {
        //Check if the current point (currentPointIndex) is different from the point being checked (j) to avoid comparing the point with itself.
        //Call the isProximityCriteriaMet function to determine if the proximity criteria are met between the points at 
        //currentPointIndex and j at time t and threshold distance D.
        if (currentPointIndex != j && isProximityCriteriaMet(&points[currentPointIndex], &points[j], t, D))
        {
            //If the proximity criteria are met, increment the count variable.
            count++;
        }
    }
    return count;
}


/*
checking the proximity of points in the points array at different time points represented by the tValues array. 
For each time point, the function iterates over all points and determines the number of points in proximity to each point based on the 
specified threshold distance D. If the number of proximity points (count) for a specific point is greater than or equal to the minimum 
required proximity points K, the point is considered a proximity point, and the results array is updated accordingly using the updateResults function.
*/
__global__ void checkProximityAtT(Point *points, double *tValues, const int tCount, const int N, const int K, const double D, int *results)
{
/*
*param points (Point*): A pointer to the array of Point structures representing the initial state of the points.
*param tValues (double*): A pointer to the array containing different time points.
*param tCount (const int): The total number of time points in the tValues array.
*param N (const int): The total number of points in the points array.
*param K (const int): The minimum number of proximity points required for a point to be considered a proximity point.
*param D (const double): The threshold distance used as the proximity criteria.
*param results (int*): A pointer to the results array.
**/
    //Calculate the index of the current thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    //Check if the calculated index is beyond the valid range of time points (tCount). -> do nothing
    if (idx >= tCount)
        return;

    //Retrieve the time value (t) corresponding to the current index from the tValues array.
    double t = tValues[idx];
    // Counter to track the number of points found for this t.
    int pointsFound = 0;

    for (int i = 0; i < N; i++)
    {
        //Call the countProximityPoints function to determine the number of points in proximity to the point at index i 
        //at time t and within the threshold distance D.
        int count = countProximityPoints(points, N, i, t, D);
        //Check if the count of proximity points (count) is greater than or equal to the minimum required proximity points K.
        if (count >= K)
        {
            //Retrieve the ID of the point at index i from the points array.
            int proximityPointId = points[i].id;
            // update the results array with the proximityPointId for the current time point index idx if the proximity criteria are met for the point at index i.
            updateResults(idx, results, proximityPointId);
            // Increment the counter for points found.
            pointsFound++; 
            // Break the loop if three points are found.
            if (pointsFound == 3)
                break;
        }
    }
}



//This function checks for any CUDA errors during GPU memory allocation and computation.
void checkCudaError(hipError_t error, Point *dPoints, double *dTValues, int *dResults)
{
/**
*param error: A hipError_t variable representing the CUDA error status.
*param dPoints: A pointer to the GPU memory for points data.
*param dTValues: A pointer to the GPU memory for tValues data.
*param dResults: A pointer to the GPU memory for results data.
**/
    if (error != hipSuccess)
    {
        fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(error));
        hipFree(dPoints);
        hipFree(dTValues);
        hipFree(dResults);
        exit(EXIT_FAILURE);
    }
}

//This function performs the proximity computation on the GPU using CUDA
void performGPUComputation(int *N, int *K, double *D, int *tCountSize, double *myTValues, Point *points, int *results)
{
    hipError_t error = hipSuccess;
    //Calculate the number of threads per block for the GPU kernel computation, ensuring that the number of threads does not exceed BLOCK_SIZE.
    int threadPerBlock = min(BLOCK_SIZE, *tCountSize);
    //Calculate the number of blocks per grid for the GPU kernel computation based on the tCountSize.
    int blocksPerGrid = (*tCountSize + threadPerBlock - 1) / threadPerBlock;

    //Declare device pointers (dPoints, dTValues, dResults) to hold the corresponding data on the GPU.
    Point *dPoints = NULL;
    double *dTValues = NULL;
    int *dResults = NULL;

    //Allocate memory on the device for dPoints, dTValues, and dResults.
    error = hipMalloc((void **)&dPoints, (*N) * sizeof(Point));
    checkCudaError(error, dPoints, dTValues, dResults);
    error = hipMalloc((void **)&dTValues, (*tCountSize) * sizeof(double));
    checkCudaError(error, dPoints, dTValues, dResults);
    error = hipMalloc((void **)&dResults, CONSTRAINTS * (*tCountSize) * sizeof(int));
    checkCudaError(error, dPoints, dTValues, dResults);

    //Copy the points data from the host to the device (dPoints) to enable GPU computations.
    error = hipMemcpy(dPoints, points, (*N) * sizeof(Point), hipMemcpyHostToDevice);
    checkCudaError(error, dPoints, dTValues, dResults);
    //Copy the tValues data from the host to the device (dTValues) to enable GPU computations.
    error = hipMemcpy(dTValues, myTValues, (*tCountSize) * sizeof(double), hipMemcpyHostToDevice);
    checkCudaError(error, dPoints, dTValues, dResults);
    //Copy the results data from the host to the device (dResults) to enable GPU computations.
    error = hipMemcpy(dResults, results, CONSTRAINTS * (*tCountSize) * sizeof(int), hipMemcpyHostToDevice);
    checkCudaError(error, dPoints, dTValues, dResults);

    // Launch the GPU kernel function checkProximityAtT with the specified grid and block configurations to perform 
    // proximity computation on the GPU.
    checkProximityAtT<<<blocksPerGrid, threadPerBlock>>>(dPoints, dTValues, *tCountSize, *N, *K, *D, dResults);

    //Synchronize the device, ensuring all GPU computations are completed before proceeding.
    error = hipDeviceSynchronize();
    checkCudaError(error, dPoints, dTValues, dResults);

    //Copy the results of the GPU computation from the device (GPU) memory to the host (CPU) memory.
    error = hipMemcpy(results, dResults, CONSTRAINTS * (*tCountSize) * sizeof(int), hipMemcpyDeviceToHost);
    checkCudaError(error, dPoints, dTValues, dResults);

    //Release the device memory that was allocated for dPoints, dTValues, and dResults. This step is crucial to 
    //avoid memory leaks and efficiently manage GPU resources.
    hipFree(dPoints);
    hipFree(dTValues);
    hipFree(dResults);
}

